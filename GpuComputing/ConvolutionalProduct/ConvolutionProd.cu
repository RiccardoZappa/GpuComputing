#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Utils\GpuTimer.cuh"

constexpr auto FILTER_RADIUS = 2;
using namespace utils;
__constant__ float f [FILTER_RADIUS * 2 + 1][FILTER_RADIUS * 2 + 1];

__global__ void convolution_2D_basic_kernel(float* N, float* P,const int r, int width, int height)
{
    int out_col = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row = blockIdx.y * blockDim.y + threadIdx.y;
    float Pvalue = 0.0f;
    for (int f_Row = 0; f_Row < 2*r + 1; f_Row++)
    {
	    for(int f_Col=0; f_Col < 2*r + 1; f_Col++)
	    {
            int in_Row = out_row - r + f_Row;
            int in_Col = out_col - r + f_Col;
            if (in_Row >= 0 && in_Row < height && in_Col >= 0 && in_Col < width)
            {
                Pvalue += f[f_Row][f_Col] * N[in_Row * width + in_Col];
            }
	    }
    }
    P[out_row * width + out_col] = Pvalue;
}

int main()
{
	GpuTimer gpu_timer;
    float* f_h;
    // create my filter f_h
    hipMemcpyToSymbol(HIP_SYMBOL(f_h), f, (FILTER_RADIUS * 2 + 1) * (FILTER_RADIUS * 2 + 1) * sizeof(float));
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    return 0;
}

