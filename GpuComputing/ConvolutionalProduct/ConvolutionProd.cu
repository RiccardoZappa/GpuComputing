#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Utils\GpuTimer.cuh"

#include <stdio.h>
#include <stdlib.h>
#include "bmpUtils.h"
#include <iostream>

constexpr auto FILTER_RADIUS = 1;
using namespace utils;
__constant__ float f [FILTER_RADIUS * 2 + 1][FILTER_RADIUS * 2 + 1];

__global__ void convolution_2D_basic_kernel(float* N, float* P,const int r, int width, int height)
{
    int out_col = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row = blockIdx.y * blockDim.y + threadIdx.y;
    float Pvalue = 0.0f;
    for (int f_Row = 0; f_Row < 2*r + 1; f_Row++)
    {
	    for(int f_Col=0; f_Col < 2*r + 1; f_Col++)
	    {
            int in_Row = out_row - r + f_Row;
            int in_Col = out_col - r + f_Col;
            if (in_Row >= 0 && in_Row < height && in_Col >= 0 && in_Col < width)
            {
                Pvalue += f[f_Row][f_Col] * N[in_Row * width + in_Col];
            }
	    }
    }
    P[out_row * width + out_col] = Pvalue;
}
/*
 *  Kernel that apply grayscale to my image
 */
__global__ void ImgGrayscale(pel* ImgDst, pel* ImgSrc, uint width) {

	int idx = blockIdx.x * 128 + threadIdx.x;
	uint BlockPerRow = (width + 127) / 128;
	uint rows = blockIdx.x / BlockPerRow;
	uint columns = idx - rows * width;
	uint numBytePerRow = (width * 3 + 3) & (~3);
	uint IndexSrc = numBytePerRow * rows + columns * 3;

	ImgDst[IndexSrc] = 0.299*ImgSrc[IndexSrc] + 0.587*ImgSrc[IndexSrc + 1] + 0.114*ImgSrc[IndexSrc + 2];
	ImgDst[IndexSrc + 1] = 0.299*ImgSrc[IndexSrc] + 0.587*ImgSrc[IndexSrc + 1] + 0.114 *ImgSrc[IndexSrc + 2];
	ImgDst[IndexSrc + 2] = 0.299*ImgSrc[IndexSrc] + 0.587*ImgSrc[IndexSrc + 1] + 0.114*ImgSrc[IndexSrc + 2];
}
/*
 *  Read a 24-bit/pixel BMP file into a 1D linear array.
 *  Allocate memory to store the 1D image and return its pointer
 */
pel* ReadBMPlin(char* fn) {
	static pel* Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL) {
		printf("\n\n%s NOT FOUND\n\n", fn);
		exit(EXIT_FAILURE);
	}

	pel HeaderInfo[54];
	size_t nByte = fread(HeaderInfo, sizeof(pel), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*)&HeaderInfo[18];
	img.width = width;
	int height = *(int*)&HeaderInfo[22];
	img.height = height;
	int RowBytes = (width * 3 + 3) & (~3);  // row is multiple of 4 pixel
	img.rowByte = RowBytes;

	//save header for re-use
	memcpy(img.headInfo, HeaderInfo, 54);
	printf("\n Input File name: %5s  (%d x %d)   File Size=%lu", fn, img.width,
		img.height, IMAGESIZE);

	// allocate memory to store the main image (1 Dimensional array)
	Img = (pel*)malloc(IMAGESIZE);
	if (Img == NULL)
		return Img;      // Cannot allocate memory
	// read the image from disk
	size_t out = fread(Img, sizeof(pel), IMAGESIZE, f);
	fclose(f);
	return Img;
}

/*
 *  Write the 1D linear-memory stored image into file
 */
void WriteBMPlin(pel* Img, char* fn) {
	FILE* f = fopen(fn, "wb");
	if (f == NULL) {
		printf("\n\nFILE CREATION ERROR: %s\n\n", fn);
		exit(1);
	}
	//write header
	fwrite(img.headInfo, sizeof(pel), 54, f);
	//write data
	fwrite(Img, sizeof(pel), IMAGESIZE, f);
	printf("\nOutput File name: %5s  (%u x %u)   File Size=%lu", fn, img.width,
		img.height, IMAGESIZE);
	fclose(f);
}

int main()
{
	uint dimBlock = 128, dimGrid;
	char fileName[100] = "C:\\GpuComputing\\GpuComputing\\images\\dog.bmp";
	char fileNameWrite[100] = "C:\\GpuComputing\\GpuComputing\\images\\dogGray.bmp";
	pel* imgSrc, * imgDst;		 // Where images are stored in CPU
	pel* imgSrcGPU, *imgDstGPU, *imgHelpGPU;	 // Where images are stored in GPU
	GpuTimer gpuTimer; // to monitor the performance of the gpu operations
	hipError_t error;
	// Create CPU memory to store the input and output images
	imgSrc = ReadBMPlin(fileName); // Read the input image if memory can be allocated
	if (imgSrc == NULL) {
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	imgDst = (pel*)malloc(IMAGESIZE);
	if (imgDst == NULL) {
		free(imgSrc);
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}
	// Allocate GPU buffer for the input and output images
	error = hipMalloc((void**)&imgSrcGPU, IMAGESIZE * sizeof(pel));
	if (error != hipSuccess)
	{
		printf("Error in CudaMalloc imgSrcGpu: %d/n", error);
		return -1;
	}

	error = hipMalloc((void**)&imgDstGPU, IMAGESIZE * sizeof(pel));
	if (error != hipSuccess)
	{
		printf("Error in CudaMalloc imgDstGpu: %d/n", error);
		return -1;
	}
	error = hipMalloc((void**)&imgHelpGPU, IMAGESIZE * sizeof(pel));
	if (error != hipSuccess)
	{
		printf("Error in CudaMalloc imgDstGpu: %d/n", error);
		return -1;
	}
	// Copy input vectors from host memory to GPU buffers.
	error = hipMemcpy(imgSrcGPU, imgSrc, IMAGESIZE, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("Error in hipMemcpy imgSrc to imgSrcGpu: %d/n", error);
		return -1;
	}

	gpuTimer.Start();
	// invoke kernels (define grid and block sizes)
	int rowBlock = (WIDTH + dimBlock - 1) / dimBlock;
	dimGrid = HEIGHT * rowBlock;

	ImgGrayscale << <dimGrid, dimBlock >> > (imgHelpGPU, imgSrcGPU, WIDTH);
	// Copy output (results) from GPU buffer to host (CPU) memory.
	hipMemcpy(imgDst, imgHelpGPU, IMAGESIZE, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	gpuTimer.Stop();
	WriteBMPlin(imgDst, fileNameWrite);
	printf("\nKernel elapsed time %f ms \n\n", gpuTimer.Elapsed());

    const float f_h[] {(0.0f), (-1.0f), (0.0f), (-1.0f), (4.0f), (-1.0f), (0.0f), (-1.0f), (0.0f)};

	// create my filter f_h
    hipMemcpyToSymbol(HIP_SYMBOL(f), f_h, (FILTER_RADIUS * 2 + 1) * (FILTER_RADIUS * 2 + 1) * sizeof(float));



	//// Copy output (results) from GPU buffer to host (CPU) memory.
	//hipMemcpy(imgDst, imgDstGPU, IMAGESIZE, hipMemcpyDeviceToHost);
	//// Write the flipped image back to disk
	//WriteBMPlin(imgDst, fileNameWrite);
	//printf("\nKernel elapsed time %f ms \n\n", gpuTimer.Elapsed());

	// Deallocate CPU, GPU memory and destroy events.

	// cuda free vars
	error = hipFree(imgSrcGPU);
	if (error != hipSuccess)
	{
		printf("Error in CudaFree imgSrcGpu: %d/n", error);
		return -1;
	}
	error = hipFree(imgDstGPU);
	if (error != hipSuccess)
	{
		printf("Error in CudaFree imgDstGpu: %d/n", error);
		return -1;
	}
	error = hipFree(imgHelpGPU);
	if (error != hipSuccess)
	{
		printf("Error in CudaFree imgDstGpu: %d/n", error);
		return -1;
	}
	free(imgSrc);
	free(imgDst);

    return 0;
}

